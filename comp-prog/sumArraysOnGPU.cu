#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                                         \
{                                                                           \
    const hipError_t error = call;                                         \    
    if (error != hipSuccess){                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                       \        
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(-10 * error);                                                            \
    }                                                                       \
}                                                                           \

void checkResult(float *hostRef, float *gpuRef, const int N){
    double epsilon = 1.0E-8;
    bool match = true;

    for(int i = 0; i < N; i++){
        if(abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = false;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if(match) printf("Arrays match\n");
}

void initialData(float *ip, int size){
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));

    for(int i = 0; i < size; i++){
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N){
    for(int i = 0; i < N; i++){
        C[i] = A[i] + B[i];
    }
}

__global__ sumArraysOnGPU(float *A, float *B, float *C, const int N){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N){
        C[i] = A[i] + B[i];
    } 
}

int main(int argc, char **argv) {
    printf("%s Starting...\n", argv[0]);

    
    int dev = 0;
    // get device properties
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("using device %d: %s\n", dev, deviceProp.name);

    // set up device
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 1<<24;  
    printf("Vector size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;

    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    // initialize data on host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    // initializing with 0    
    memset(hostRef, 0, nElem);
    memset(gpuRef, 0, nElem);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, nBytes);
    hipMalloc((void**)&d_B, nBytes);
    hipMalloc((void**)&d_C, nBytes);

    // transfer data from host to devie memory
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // invoke kernel at the host side
    dim3 block (256);
    dim3 grid  ((nElem + block.x - 1)/block.x);

    sumArraysOnGPU <<< grid, block >>> (d_A, d_B, d_C, N);
    printf("Execution Configuration <<< %d, %d >>> \n", grid.x, block.x);

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_A, nBytes, hipMemcpyDeviceToHost);

    // add vectors on host to compare the result
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    
    // check results
    checkResult(hostRef, gpuRef, nElem);
    
    // free device gloabl memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);
     
    return(0); 
}


/* Summing Matrices with a 2D Grid and 2D Blocks
 // Launch Configurations

dim3 block(32, 32);
dim3 grid((nCols + block.x - 1)/block.x, nRows + block.y - 1)/block.y);
*/
__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nRows, int nCols){
    unsigned int ix = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int iy = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int idx = iy*nCols + ix;

    if(ix < nCols && iy < nRows){
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}


/* Summing Matrices with a 1D Grid and 1D Blocks
// Launch Configurations

dim3 block(32,1);
dim3 grid((nCols + block.x - 1)/block.x, 1);
*/

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nRows, int nCols){
    unsigned int ix = threadIdx.x + blockDim.x * blockIdx.x;
    if(ix < nCols){
        for( int iy = 0; iy < nRows; iy++) {
            unsigned int idx = iy*nCols + ix;
            MatC[idx] = MatA[idx] + MatB[idx];
        }
    }
}


/* Summing Matrices with a 2D Grid and 1D Blocks
// Launch Configuration 

dim3 block(32);
dim3 grid((nCols + block.x -1) /block.x, nRows);
*/

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nRows, int nCols){
    unsigned int ix = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int iy = blockIdx.y;
    unsigned int idx = iy * nCols + ix;
    if (ix < nCols){
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}


/*

$ nvcc sumArraysOnGPU.cu  -o addvector
$ ./addvector

** Timing with CPU Timer **
#include <sys/time.h>

double iStart, iElaps;
iStart = cpuSecond();
// some operation or function call
iElaps = cpuSecond() - iStart();

printf("Time Elapsed %f\n", iElaps);

** Timing with `nvprof`
$ nvprof --help
$ nvprof ./addvector

*/